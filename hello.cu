
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello from CUDA kernel!\n");
}

int main() {
    hello_cuda<<<1, 1>>>(); // Launch kernel
    hipDeviceSynchronize(); // Ensure kernel execution completes
    return 0;
}
